#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

/* Functia care returneaza informatiile necesare rularii kernel-ului */

hipError_t getGPUInformation(int &numBlocks, int &numThreads,
							  int numItems)
{
	hipDeviceProp_t deviceProp;
	hipError_t returnValue;

	returnValue = hipGetDeviceProperties(&deviceProp, 0);
	DIE(returnValue, "hipGetDeviceProperties() failed");

	numThreads = deviceProp.maxThreadsPerBlock;
	numBlocks = numItems / numThreads;

	if (numBlocks * numThreads != numItems)
	{
		numBlocks = numBlocks + 1;
	}

	return (hipError_t)0;
}

/* Metode specifice clasei care modeleaza Hash Table-ul */

HashTable::HashTable(Entry *entries, int capacity, int size)
{
	this->entries = entries;
	this->capacity = capacity;
	this->size = size;
}

HashTable::HashTable()
{
	this->capacity = 0;
	this->size = 0;
	this->entries = nullptr;
}

/* Funtia care modeleaza functia de dispersie aproape injectiva */

static __device__ int triple32inc(long long x, int capacity)
{
	long long var = x * PRODUCT;
	var = var % MODULO;
	return var % capacity;
}

/* Functia in care kernel-ul recalculeaza cheia din bucket-urile vechi si
   o plaseaza impreuna cu valoarea sa in noul set de bucket-uri */

__global__ void reshape_entry(Entry *oldEntries, int oldCapacity,
							  Entry *newEntries, int newCapacity)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= oldCapacity)
		return;
	if (oldEntries[idx].key == KEY_INVALID)
		return;

	int key_to_add = oldEntries[idx].key;
	int hash = triple32inc(key_to_add, newCapacity);

	while (1)
	{
		int key_before = atomicCAS(&newEntries[hash].key,
								   KEY_INVALID, key_to_add);
		if (key_before == KEY_INVALID)
		{
			atomicExch(&newEntries[hash].value, oldEntries[idx].value);
			return;
		}
		hash = (hash + 1) % newCapacity;
	}
}

/* Functia in care kernel-ul cauta sa puna in vectorul
   'values' valoarea corespunzatoare cheii */

__global__ void get_entry(int *keys, int *values, int numKeys,
						  HashTable hashTable)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= numKeys)
		return;
	int key_to_find = keys[idx];
	int hash = triple32inc(key_to_find, hashTable.capacity);

	while (1)
	{
		if (hashTable.entries[hash].key == key_to_find)
		{
			atomicExch(&values[idx], hashTable.entries[hash].value);
			return;
		}
		hash = (hash + 1) % hashTable.capacity;
	}
}

/* Functia in care kernel-ul se ocupa cu inserarea unui
   singur element in Hash Table folosind Linear Probing */

__global__ void insert_entry(int *keys, int *values, int numKeys,
							 HashTable hashTable)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= numKeys)
		return;
	if (keys[idx] <= 0 || values[idx] <= 0)
		return;

	int addKeys = keys[idx];
	int hash = triple32inc(addKeys, hashTable.capacity);

	while (1)
	{
		int oldKey = atomicCAS(&hashTable.entries[hash].key,
							   KEY_INVALID, addKeys);

		/* Valoarea se modifica atunci cand 'oldKey' este 'KEY_INVALID'
		   sau cand este identica cu noua cheie */

		if (oldKey == KEY_INVALID || oldKey == addKeys)
		{
			atomicExch(&hashTable.entries[hash].value, values[idx]);
			return;
		}
		hash = (hash + 1) % hashTable.capacity;
	}
}

/* Metodele de prelucrare pentru un Hash Table pe GPU */

GpuHashTable::GpuHashTable(int size)
{
	hipError_t error;
	hashTable.entries = nullptr;
	hashTable.size = 0;
	hashTable.capacity = size;

	error = glbGpuAllocator->_cudaMalloc((void **)&hashTable.entries, size * sizeof(Entry));
	DIE(error, "hipMalloc() failed");
	error = hipMemset(hashTable.entries, 0, size * sizeof(Entry));
	DIE(error, "hipMemset() failed");
}

/* Stergere Hash Table */

GpuHashTable::~GpuHashTable()
{
	DIE(glbGpuAllocator->_cudaFree(hashTable.entries), "hipFree() failed");
}

/* Redimensionare Hash Table */

void GpuHashTable::reshape(int numBucketsReshape)
{
	Entry *entries;
	hipError_t error;
	int numBlocks = 0, numThreads = 0;

	error = glbGpuAllocator->_cudaMalloc((void **)&entries, numBucketsReshape * sizeof(Entry));
	DIE(error, hipGetErrorString(error));
	error = hipMemset(entries, 0, numBucketsReshape * sizeof(Entry));
	DIE(error, hipGetErrorString(error));

	getGPUInformation(numBlocks, numThreads, hashTable.capacity);
	reshape_entry<<<numBlocks, numThreads>>>(hashTable.entries,
											 hashTable.capacity, entries, numBucketsReshape);

	error = hipDeviceSynchronize();
	DIE(error, hipGetErrorString(error));

	error = glbGpuAllocator->_cudaFree(hashTable.entries);
	DIE(error, hipGetErrorString(error));

	hashTable.entries = entries;
	hashTable.capacity = numBucketsReshape;
}

/* Inserare Hash Table */

bool GpuHashTable::insertBatch(int *keys, int *values, int numKeys)
{
	/* Se modifica dimensiunea cand se depaseste pragul maxim */

	if (static_cast<float>(hashTable.size + numKeys) /
			static_cast<float>(hashTable.capacity) >
		MAX_LOAD_LIMIT)
	{

		reshape(static_cast<float>(hashTable.size +
								   numKeys) /
				MIN_LOAD_LIMIT);
	}

	int *gpuKeys, *gpuValues;
	hipError_t error;
	int numBlocks, numThreads;

	error = glbGpuAllocator->_cudaMalloc((void **)&gpuKeys, numKeys * sizeof(int));
	DIE(error, hipGetErrorString(error));
	error = glbGpuAllocator->_cudaMalloc((void **)&gpuValues, numKeys * sizeof(int));
	DIE(error, hipGetErrorString(error));

	error = hipMemcpy(gpuKeys, keys, numKeys * sizeof(int),
					   hipMemcpyHostToDevice);
	DIE(error, hipGetErrorString(error));
	error = hipMemcpy(gpuValues, values, numKeys * sizeof(int),
					   hipMemcpyHostToDevice);
	DIE(error, hipGetErrorString(error));

	getGPUInformation(numBlocks, numThreads, numKeys);
	insert_entry<<<numBlocks, numThreads>>>(gpuKeys, gpuValues,
											numKeys, hashTable);

	error = hipDeviceSynchronize();
	DIE(error, hipGetErrorString(error));
	hashTable.size += numKeys;

	error = glbGpuAllocator->_cudaFree(gpuKeys);
	DIE(error, hipGetErrorString(error));
	error = glbGpuAllocator->_cudaFree(gpuValues);
	DIE(error, hipGetErrorString(error));

	return true;
}

/* Obtinerea unor elemente din Hash Table */

int *GpuHashTable::getBatch(int *keys, int numKeys)
{
	int *gpuKeys, *gpuValues, *result;
	hipError_t error;
	int numBlocks, numThreads;

	result = (int *)malloc(numKeys * sizeof(int));
	DIE(!result, "Malloc");
	error = glbGpuAllocator->_cudaMalloc((void **)&gpuKeys, numKeys * sizeof(int));
	DIE(error, hipGetErrorString(error));
	error = glbGpuAllocator->_cudaMalloc((void **)&gpuValues, numKeys * sizeof(int));
	DIE(error, hipGetErrorString(error));

	error = hipMemset(gpuValues, -1, numKeys * sizeof(int));
	DIE(error, hipGetErrorString(error));
	error = hipMemcpy(gpuKeys, keys, numKeys * sizeof(int),
					   hipMemcpyHostToDevice);
	DIE(error, hipGetErrorString(error));

	getGPUInformation(numBlocks, numThreads, hashTable.capacity);
	get_entry<<<numBlocks, numThreads>>>(gpuKeys, gpuValues,
										 numKeys, hashTable);

	error = hipDeviceSynchronize();
	DIE(error, hipGetErrorString(error));

	error = hipMemcpy((void **)result, gpuValues, numKeys * sizeof(int),
					   hipMemcpyDeviceToHost);
	DIE(error, hipGetErrorString(error));

	error = glbGpuAllocator->_cudaFree(gpuKeys);
	DIE(error, hipGetErrorString(error));
	error = glbGpuAllocator->_cudaFree(gpuValues);
	DIE(error, hipGetErrorString(error));

	return result;
}
